#include "hip/hip_runtime.h"
#define FLOAT_TO_BITS(x) (*reinterpret_cast<unsigned int*>(x))
#define BITS_TO_FLOAT(x) (*reinterpret_cast<float*>(x))

__device__ __forceinline__ unsigned int extract_exponent(float *a) {
  unsigned int temp = *(reinterpret_cast<unsigned int*>(a));
  temp = (temp << 1 >> 24); // single preciision, 1 sign bit, 23 mantissa bits
  return temp-127+1; // exponent offset and virtual bit
}

__device__ __forceinline__ unsigned int clip_exponent(int exp_bits, int man_bits,
                                                      unsigned int old_num,
                                                      unsigned int quantized_num) {
  if (quantized_num == 0)
    return quantized_num;

  int quantized_exponent_store = quantized_num << 1 >> 1 >> 23; // 1 sign bit, 23 mantissa bits
  int max_exponent_store = (1 << (exp_bits - 1)) + 127; // we are not reserving an exponent bit for infinity, nan, etc
  // Clippping Value Up
  if (quantized_exponent_store > max_exponent_store)
  {
    unsigned int max_man = (unsigned int)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits); // 1 sign bit, 8 exponent bits, 1 virtual bit
    unsigned int max_num = ((unsigned int)max_exponent_store << 23) | max_man;
    unsigned int old_sign = old_num >> 31 << 31;
    quantized_num = old_sign | max_num;
  }
  return quantized_num;
}


__device__ __forceinline__ unsigned int clip_max_exponent(int man_bits,
                                                          unsigned int max_exponent,
                                                          unsigned int quantized_num) {
  unsigned int quantized_exponent = quantized_num << 1 >> 24 << 23; // 1 sign bit, 23 mantissa bits
  if (quantized_exponent > max_exponent) {
    unsigned int max_man = (unsigned int ) -1 << 9 >> 9 >> (23-man_bits) << (23-man_bits); // 1 sign bit, 8 exponent bits
    unsigned int max_num = max_exponent | max_man;
    unsigned int old_sign = quantized_num >> 31 << 31;
    quantized_num = old_sign | max_num;
  }
  return quantized_num;
}

Tensor get_max_entry(Tensor a, int dim) {
  Tensor max_entry;
  if (dim == -1) {
    max_entry = at::max(at::abs(a)).expand_as(a).contiguous();
  } else if (dim == 0) {
    Tensor input_view = a.view({a.size(0), -1});
    max_entry = std::get<0>(input_view.abs().max(1, true)).expand_as(input_view).view_as(a).contiguous();
  } else {
    Tensor input_transpose = a.transpose(0, dim);
    Tensor input_view = input_transpose.contiguous().view({input_transpose.size(0), -1});
    Tensor max_transpose = std::get<0>(input_view.abs().max(1, true)).expand_as(input_view).view_as(input_transpose);
    max_entry = max_transpose.transpose(dim, 0).contiguous();
  }
  return max_entry;
}